#include "hip/hip_runtime.h"
// **************************************************
// Copyright (c) 2025, Mayank Mishra
// **************************************************

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "include/xma.h"

namespace ck = flash_model_architectures;
namespace ck_mem = ck::memory;

using fp32 = ck::fp32;
using uint32 = ck::uint32;
using uint64 = ck::uint64;

template <typename scalar_t>
inline __device__ void _swiglu_backward(const scalar_t &gate,
                                        const scalar_t &up,
                                        const scalar_t &output_grad,
                                        scalar_t *gate_grad_buffer,
                                        scalar_t *up_grad_buffer,
                                        const uint32 &index) {
    using dtype = ck::DType<scalar_t>;

    fp32 _gate = dtype::upcast(gate);
    fp32 _up = dtype::upcast(up);
    fp32 _output_grad = dtype::upcast(output_grad);

    fp32 _gate_sigmoid = ck::sigmoid<fp32, fp32>(_gate);
    fp32 _gate_silu = _gate * _gate_sigmoid;

    fp32 _gate_grad = _output_grad * _up * (_gate_sigmoid + _gate_silu * (1 - _gate_sigmoid));
    fp32 _up_grad = _output_grad * _gate_silu;

    scalar_t gate_grad = dtype::downcast(_gate_grad);
    scalar_t up_grad = dtype::downcast(_up_grad);

    gate_grad_buffer[index] = gate_grad;
    up_grad_buffer[index] = up_grad;
}

template <typename scalar_t>
__global__ void swiglu_backward_cuda_kernel(const scalar_t *gate,
                                            const scalar_t *up,
                                            const scalar_t *output_grad,
                                            scalar_t *gate_grad,
                                            scalar_t *up_grad,
                                            const uint64 N) {
    constexpr uint32 N_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();

    const uint32 thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32 N_vec = N / N_per_thread;

    if (thread_id < N_vec) {
        const scalar_t *gate_vec = ck_mem::load_128_bits<scalar_t>(gate, thread_id);
        const scalar_t *up_vec = ck_mem::load_128_bits<scalar_t>(up, thread_id);
        const scalar_t *output_grad_vec = ck_mem::load_128_bits<scalar_t>(output_grad, thread_id);

        scalar_t gate_grad_buffer[N_per_thread];
        scalar_t up_grad_buffer[N_per_thread];

        for (uint32 i = 0; i < N_per_thread; i++) {
            _swiglu_backward<scalar_t>(
                gate_vec[i], up_vec[i], output_grad_vec[i], gate_grad_buffer, up_grad_buffer, i);
        }

        ck_mem::store_128_bits<scalar_t>(gate_grad_buffer, gate_grad, thread_id);
        ck_mem::store_128_bits<scalar_t>(up_grad_buffer, up_grad, thread_id);
    }

    const uint32 index = N_vec * N_per_thread + thread_id;
    if (index < N) {
        _swiglu_backward<scalar_t>(gate[index], up[index], output_grad[index], gate_grad, up_grad, index);
    }
}

void swiglu_backward_cuda(const torch::Tensor &gate,
                          const torch::Tensor &up,
                          const torch::Tensor &output_grad,
                          torch::Tensor &gate_grad,
                          torch::Tensor &up_grad,
                          const uint32 &BLOCK_SIZE) {
    CHECK_CUDA_TENSOR(gate);
    CHECK_CUDA_TENSOR(up);
    CHECK_CUDA_TENSOR(output_grad);
    CHECK_CUDA_TENSOR(gate_grad);
    CHECK_CUDA_TENSOR(up_grad);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE);

    const uint64 total_elements = gate.numel();

    DISPATCH_FLOAT_KERNEL(gate.scalar_type(), "swiglu_backward_cuda_kernel", scalar_t, ([&] {
                              const uint32 N_per_thread = ck_mem::get_num_elements_for_vector_load_stores<scalar_t>();
                              const uint32 N_per_block = BLOCK_SIZE * N_per_thread;

                              std::vector<ck::ChunkedArray<scalar_t>> gate_chunks =
                                  ck::chunk_array<scalar_t>(gate.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> up_chunks =
                                  ck::chunk_array<scalar_t>(up.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> output_grad_chunks =
                                  ck::chunk_array<scalar_t>(output_grad.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> gate_grad_chunks =
                                  ck::chunk_array<scalar_t>(gate_grad.data_ptr<scalar_t>(), total_elements);
                              std::vector<ck::ChunkedArray<scalar_t>> up_grad_chunks =
                                  ck::chunk_array<scalar_t>(up_grad.data_ptr<scalar_t>(), total_elements);

                              for (uint32 i = 0; i < gate_chunks.size(); i++) {
                                  ck::ChunkedArray<scalar_t> gate_chunk = gate_chunks[i];
                                  ck::ChunkedArray<scalar_t> up_chunk = up_chunks[i];
                                  ck::ChunkedArray<scalar_t> output_grad_chunk = output_grad_chunks[i];
                                  ck::ChunkedArray<scalar_t> gate_grad_chunk = gate_grad_chunks[i];
                                  ck::ChunkedArray<scalar_t> up_grad_chunk = up_grad_chunks[i];

                                  const uint64 N = gate_chunk.num_elements;
                                  const uint32 NUM_BLOCKS = ck::ceil_divide<uint64>(N, N_per_block);

                                  swiglu_backward_cuda_kernel<scalar_t>
                                      <<<NUM_BLOCKS, BLOCK_SIZE>>>(gate_chunk.array,
                                                                   up_chunk.array,
                                                                   output_grad_chunk.array,
                                                                   gate_grad_chunk.array,
                                                                   up_grad_chunk.array,
                                                                   N);
                              }
                          }));
}
